#include "hip/hip_runtime.h"
#ifndef CUDA_TILE_SPACE_LIGHT_GRID
#define CUDA_TILE_SPACE_LIGHT_GRID

#include "cuda_tile_space_light_grid.cuh"
#define COMPILE_FOR_CUDA 1
#include "AxisAlignedBoundingBox.h"
#include "Vector2.h"
#include "Matrix4.h"
#include "Camera.h"
#include "CameraFwdDeclare.h"
#include "ScreenRect.h"


//using namespace OpenIG::Library::Graphics;

__global__
void _update_grid_counts(
   int* m_GridOffsetsAndCountsCUDA
 , const AxisAlignedBoundingBox_64* pLightWorldAABBs
 , unsigned int viewport_width, unsigned int viewport_height
 , const double _matViewProjection[16]
 , unsigned int tileSizeX, unsigned int tileSizeY
 , unsigned int rejectAreaW, unsigned int rejectAreaH
 , unsigned int tileGridMaxDimsX, unsigned int tileGridMaxDimsY
 , size_t numLights)
{
   int lightIndex = blockDim.x * blockIdx.x + threadIdx.x;

   if (lightIndex<numLights)
   {
      Vector2_uint32 viewPortSize(viewport_width, viewport_height);
      Vector2_uint32 m_TileSize(tileSizeX, tileSizeY);
      Vector2_uint32 rejectArea(rejectAreaW, rejectAreaH);
      
      ScreenRect rect = Camera_64::GetScreenAABB(pLightWorldAABBs[lightIndex], viewPortSize, Matrix4_64(_matViewProjection));

      // Culled
      if (rect.width() < rejectArea.x && rect.height() < rejectArea.y)
      {
         return;
      }

      Vector2_uint32 tileGridMaxDims(tileGridMaxDimsX, tileGridMaxDimsY);

      Vector2_uint32 tileLowerBound = Math::Clamp(rect.vMin / m_TileSize, Vector2_uint32(0,0), tileGridMaxDims + 1);
      Vector2_uint32 tileUpperBound = Math::Clamp((rect.vMax + m_TileSize - 1) / m_TileSize, Vector2_uint32(0,0), tileGridMaxDims + 1);

      // For each light find whether its spans a tile
      // Update grid count if so
      for (uint32 y = tileLowerBound.y; y < tileUpperBound.y; ++y)
      {
         for (uint32 x = tileLowerBound.x; x < tileUpperBound.x; ++x)
         {
            int gridCountIndex = (x + y * tileGridMaxDims.x)*2;
            atomicAdd(&m_GridOffsetsAndCountsCUDA[gridCountIndex],1);
         }
      }
   }
}

__global__
void _update_grid_offsets(int* m_GridOffsetsAndCountsCUDA
, unsigned int viewport_width, unsigned int viewport_height
, unsigned int tileSizeX, unsigned int tileSizeY
, unsigned int tileGridMaxDimsX, unsigned int tileGridMaxDimsY
)
{
   Vector2_uint32 viewPortSize(viewport_width, viewport_height);
   Vector2_uint32 m_TileSize(tileSizeX, tileSizeY);
   Vector2_uint32 tileGridMaxDims(tileGridMaxDimsX, tileGridMaxDimsY);

   uint32 offset = 0;
   for (uint32 y = 0; y < tileGridMaxDims.y; ++y)
   {
      for (uint32 x = 0; x < tileGridMaxDims.x; ++x)
      {
         int index = (x + y * tileGridMaxDims.x)*2;

         uint32 count = m_GridOffsetsAndCountsCUDA[index];
         // set offset to be just past end, then decrement while filling in
         m_GridOffsetsAndCountsCUDA[index+1] = offset + count;
         offset += count;
      }
   }
}

__global__
void _update_light_index_list_and_offsets(int* m_GridOffsetsAndCountsCUDA
, int* m_TileLightIndexListsCUDA
, const AxisAlignedBoundingBox_64* pLightWorldAABBs
, unsigned int viewport_width, unsigned int viewport_height
, const double _matViewProjection[16]
, unsigned int tileSizeX, unsigned int tileSizeY
, unsigned int rejectAreaW, unsigned int rejectAreaH
, unsigned int tileGridMaxDimsX, unsigned int tileGridMaxDimsY
, int numLights
)
{
   int lightIndex = blockDim.x * blockIdx.x + threadIdx.x;

   if (lightIndex<numLights)
   {
      Vector2_uint32 viewPortSize(viewport_width, viewport_height);
      Vector2_uint32 m_TileSize(tileSizeX, tileSizeY);
      Vector2_uint32 rejectArea(rejectAreaW, rejectAreaH);

      ScreenRect rect = Camera_64::GetScreenAABB(pLightWorldAABBs[lightIndex], viewPortSize, Matrix4_64(_matViewProjection));


      // Culled
      if (rect.width() < rejectArea.x && rect.height() < rejectArea.y)
      {
         return;
      }


      Vector2_uint32 tileGridMaxDims(tileGridMaxDimsX, tileGridMaxDimsY);

      Vector2_uint32 tileLowerBound = Math::Clamp(rect.vMin / m_TileSize, Vector2_uint32(0,0), tileGridMaxDims + 1);
      Vector2_uint32 tileUpperBound = Math::Clamp((rect.vMax + m_TileSize - 1) / m_TileSize, Vector2_uint32(0,0), tileGridMaxDims + 1);

      // For all the tiles it spans
      for (uint32 y = tileLowerBound.y; y < tileUpperBound.y; ++y)
      {
         for (uint32 x = tileLowerBound.x; x < tileUpperBound.x; ++x)
         {
               uint32 gridOffsetIndex = (x + y * tileGridMaxDims.x)*2+1;
               // store reversely into next free slot

               uint32 offset = atomicSub(&m_GridOffsetsAndCountsCUDA[gridOffsetIndex],1);
               m_TileLightIndexListsCUDA[offset-1] = lightIndex;
         }
      }
   }
}


extern "C"
void update_grid_counts_offsets(
   int* m_GridOffsetsAndCountsCUDA
 , int* m_TileLightIndexListsCUDA
 , const AxisAlignedBoundingBox_64* pLightWorldAABBs
 , unsigned int viewport_width, unsigned int viewport_height
 , const double _matViewProjection[16]
 , unsigned int tileSizeX, unsigned int tileSizeY
 , unsigned int rejectAreaW, unsigned int rejectAreaH
 , unsigned int tileGridMaxDimsX, unsigned int tileGridMaxDimsY
 , size_t numLights)
{
      // PPP: How to compute this
   int threadsPerBlock = 512;
   int blocksPerGrid = (numLights + threadsPerBlock - 1)/threadsPerBlock;

   _update_grid_counts<<<blocksPerGrid, threadsPerBlock>>>
      ( m_GridOffsetsAndCountsCUDA
      , pLightWorldAABBs
      , viewport_width, viewport_height
      , _matViewProjection
      , tileSizeX, tileSizeY
      , rejectAreaW, rejectAreaH
	  , tileGridMaxDimsX, tileGridMaxDimsY
      , numLights);

   // PPP: Could actually parallize this
   _update_grid_offsets<<<1, 1>>>(m_GridOffsetsAndCountsCUDA
      , viewport_width, viewport_height
      , tileSizeX, tileSizeY
	  , tileGridMaxDimsX, tileGridMaxDimsY);

   _update_light_index_list_and_offsets<<<blocksPerGrid, threadsPerBlock>>>
      ( m_GridOffsetsAndCountsCUDA
      , m_TileLightIndexListsCUDA
	  , pLightWorldAABBs
      , viewport_width, viewport_height
	  , _matViewProjection
      , tileSizeX, tileSizeY
      , rejectAreaW, rejectAreaH
	  , tileGridMaxDimsX, tileGridMaxDimsY
      , numLights
      );
}


#endif